#include "hip/hip_runtime.h"
//
// Created by jiashuai on 18-1-18.
//
#include <thundergbm/tree.h>
#include <thundergbm/dataset.h>
#include <thundergbm/syncmem.h>
#include <thundergbm/trainer.h>
#include "gtest/gtest.h"

//extern GBMParam global_test_param;
//
//class UpdaterTest : public ::testing::Test {
//public:
//
//    GBMParam param = global_test_param;
//
//    void SetUp() override {
//        if (!param.verbose) {
//            el::Loggers::reconfigureAllLoggers(el::Level::Debug, el::ConfigurationType::Enabled, "false");
//            el::Loggers::reconfigureAllLoggers(el::Level::Trace, el::ConfigurationType::Enabled, "false");
//        }
//        el::Loggers::reconfigureAllLoggers(el::ConfigurationType::PerformanceTracking, "false");
//    }
//
//    void TearDown() {
//        SyncMem::clear_cache();
//    }
//
//    float_type train_exact(GBMParam &param) {
//        TreeTrainer trainer;
//        return trainer.train_exact(param);
//    }
//
//    float_type train_hist(GBMParam &param) {
//        TreeTrainer trainer;
//        return trainer.train_hist(param);
//    }
//};
//
//class Exact : public UpdaterTest {
//};
//
//class Hist : public UpdaterTest {
//};
//
//TEST_F(UpdaterTest, news20_40_trees_same_as_xgboost) {
//    param.path = DATASET_DIR "news20.scale";
//    float_type rmse = train_exact(param);//5375 ms
//    EXPECT_NEAR(rmse, 2.55275, 1e-5);
//}
//
//TEST_F(UpdaterTest, abalone_40_trees_same_as_xgboost) {
//    param.path = DATASET_DIR "abalone";
//    float_type rmse = train_exact(param);//1674 ms
//    EXPECT_NEAR(rmse, 0.803684, 1e-5);
//}
//
//TEST_F(UpdaterTest, iris) {
//    param.n_trees = 2;
//    param.path = DATASET_DIR "iris.scale";
//    train_hist(param);
//}
//
//TEST_F(UpdaterTest, iris_exact) {
//    param.n_trees = 2;
//    param.path = DATASET_DIR "iris.scale";
//    train_exact(param);
//}
//
//TEST_F(Exact, covtype) {
//    param.path = DATASET_DIR "covtype";
//    train_exact(param);
//}
//
//TEST_F(Exact, e2006) {
//    param.path = DATASET_DIR "E2006.train";
//    train_exact(param);
//}
//
//TEST_F(Exact, higgs) {
//    param.path = DATASET_DIR "HIGGS";
//    train_exact(param);
//}
//
//TEST_F(Exact, ins) {
//    param.path = DATASET_DIR "ins.libsvm";
//    train_exact(param);
//}
//
//TEST_F(Exact, log1p) {
//    param.path = DATASET_DIR "log1p.E2006.train";
//    train_exact(param);
//}
//
//
//TEST_F(Exact, news20) {
//    param.path = DATASET_DIR "news20.binary";
//    train_exact(param);
//}
//
//TEST_F(Exact, real_sim) {
//    param.path = DATASET_DIR "real-sim";
//    train_exact(param);
//}
//
//TEST_F(Exact, susy) {
//    param.path = DATASET_DIR "SUSY";
//    train_exact(param);
//}
//
//TEST_F(Hist, covtype) {
//    param.path = DATASET_DIR "covtype";
//    train_hist(param);
//}
//
//TEST_F(Hist, higgs) {
//    param.path = DATASET_DIR "HIGGS";
//    train_hist(param);
//}
//
//TEST_F(Hist, ins) {
//    param.path = DATASET_DIR "ins.libsvm";
//    train_hist(param);
//}
//
//TEST_F(Hist, susy) {
//    param.path = DATASET_DIR "SUSY";
//    train_hist(param);
//}
//
//TEST_F(Hist, any) {
//    train_hist(param);
//}
