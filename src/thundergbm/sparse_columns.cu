//
// Created by shijiashuai on 5/7/18.
//
#include <thundergbm/util/cub_wrapper.h>
#include <thundergbm/sparse_columns.h>

#include "thundergbm/sparse_columns.h"
#include "thundergbm/util/device_lambda.cuh"
#include "hipsparse.h"
#include "thundergbm/util/multi_device.h"

void SparseColumns::from_dataset(const DataSet &dataset) {
    this->column_offset = 0;
    LOG(TRACE) << "constructing sparse columns, converting csr to csc";
    //cpu transpose
    n_column = dataset.n_features();
    n_row = dataset.n_instances();
    nnz = dataset.csr_val.size();
    csc_val.resize(nnz);
    csc_row_idx.resize(nnz);
    csc_col_ptr.resize(n_column + 1);
    LOG(INFO) << string_format("#non-zeros = %ld, density = %.2f%%", nnz,
                               (float) nnz / n_column / dataset.n_instances() * 100);
    auto csc_val_data = csc_val.host_data();
    auto csc_row_idx_data = csc_row_idx.host_data();
    auto csc_col_ptr_data = csc_col_ptr.host_data();
    for (int i = 0; i < nnz; ++i) {
        csc_col_ptr_data[dataset.csr_col_idx[i] + 1] += 1;
    }
    for (int i = 1; i < n_column + 1; ++i) {
        csc_col_ptr_data[i] += csc_col_ptr_data[i - 1];
    }
    for (int row = 0; row < dataset.n_instances(); ++row) {
        for (int j = dataset.csr_row_ptr[row]; j < dataset.csr_row_ptr[row + 1]; ++j) {
            int col = dataset.csr_col_idx[j]; // csr col
            int dest = csc_col_ptr_data[col]; // destination index in csc array
            csc_val_data[dest] = dataset.csr_val[j];
            csc_row_idx_data[dest] = row;
            csc_col_ptr_data[col]++; //increment column start position
        }
    }
    //recover column start position
    for (int i = 0, last = 0; i < n_column; ++i) {
        int next_last = csc_col_ptr_data[i];
        csc_col_ptr_data[i] = last;
        last = next_last;
    }

//    LOG(INFO) << "copy csr matrix to GPU";
//    //three arrays (on GPU/CPU) for csr representation
//    SyncArray<float_type> val;
//    SyncArray<int> col_idx;
//    SyncArray<int> row_ptr;
//    val.resize(dataset.csr_val.size());
//    col_idx.resize(dataset.csr_col_idx.size());
//    row_ptr.resize(dataset.csr_row_ptr.size());
//
//    //copy data to the three arrays
//    val.copy_from(dataset.csr_val.data(), val.size());
//    col_idx.copy_from(dataset.csr_col_idx.data(), col_idx.size());
//    row_ptr.copy_from(dataset.csr_row_ptr.data(), row_ptr.size());
//    LOG(INFO) << "converting csr matrix to csc matrix";
//    hipsparseHandle_t handle;
//    hipsparseMatDescr_t descr;
//    hipsparseCreate(&handle);
//    hipsparseCreateMatDescr(&descr);
//    hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);
//    hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
//
//    n_column = dataset.n_features_;
//    nnz = dataset.csr_val.size();
//    csc_val.resize(nnz);
//    csc_row_idx.resize(nnz);
//    csc_col_ptr.resize(n_column + 1);
//
//    hipsparseScsr2csc(handle, dataset.n_instances(), n_column, nnz, val.device_data(), row_ptr.device_data(),
//                     col_idx.device_data(), csc_val.device_data(), csc_row_idx.device_data(), csc_col_ptr.device_data(),
//                     HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO);
//    hipDeviceSynchronize();
//    hipsparseDestroy(handle);
//    hipsparseDestroyMatDescr(descr);
//    LOG(INFO)<<csc_val;
//    LOG(INFO)<<csc_row_idx;
//    LOG(INFO)<<csc_col_ptr;
}


//FIXME remove this function
void correct_start(int *csc_col_ptr_2d_data, int first_col_start, int n_column_sub){
    device_loop(n_column_sub + 1, [=] __device__(int col_id) {
        csc_col_ptr_2d_data[col_id] = csc_col_ptr_2d_data[col_id] - first_col_start;
    });
};
void SparseColumns::to_multi_devices(vector<std::unique_ptr<SparseColumns>> &v_columns) const {
    //devide data into multiple devices
    int n_device = v_columns.size();
    int ave_n_columns = n_column / n_device;
    DO_ON_MULTI_DEVICES(n_device, [&](int device_id) {
        SparseColumns &columns = *v_columns[device_id];
        const int *csc_col_ptr_data = csc_col_ptr.host_data();
        int first_col_id = device_id * ave_n_columns;
        int n_column_sub = (device_id < n_device - 1) ? ave_n_columns : n_column - first_col_id;
        int first_col_start = csc_col_ptr_data[first_col_id];
        int nnz_sub = (device_id < n_device - 1) ?
                      (csc_col_ptr_data[(device_id + 1) * ave_n_columns] - first_col_start) : (nnz -
                                                                                               first_col_start);
        columns.column_offset = first_col_id + this->column_offset;
        columns.nnz = nnz_sub;
        columns.n_column = n_column_sub;
        columns.n_row = n_row;
        columns.csc_val.resize(nnz_sub);
        columns.csc_row_idx.resize(nnz_sub);
        columns.csc_col_ptr.resize(n_column_sub + 1);

        columns.csc_val.copy_from(csc_val.host_data() + first_col_start, nnz_sub);
        columns.csc_row_idx.copy_from(csc_row_idx.host_data() + first_col_start, nnz_sub);
        columns.csc_col_ptr.copy_from(csc_col_ptr.host_data() + first_col_id, n_column_sub + 1);

        int *csc_col_ptr_2d_data = columns.csc_col_ptr.device_data();
        correct_start(csc_col_ptr_2d_data, first_col_start, n_column_sub);
        //correct segment start positions
        LOG(TRACE) << "sorting feature values (multi-device)";
        cub_seg_sort_by_key(columns.csc_val, columns.csc_row_idx, columns.csc_col_ptr, false);
    });
    LOG(TRACE) << "sorting finished";
}

