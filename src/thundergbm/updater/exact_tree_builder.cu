#include "hip/hip_runtime.h"
//
// Created by ss on 19-1-20.
//
#include "thundergbm/updater/exact_tree_builder.h"

#include "thundergbm/util/cub_wrapper.h"
#include "thundergbm/util/device_lambda.cuh"
#include "thrust/iterator/counting_iterator.h"
#include "thrust/iterator/transform_iterator.h"
#include "thrust/iterator/discard_iterator.h"
#include "thrust/sequence.h"
#include "thrust/binary_search.h"

void ExactTreeBuilder::InternalShard::find_split(int level) {
    TIMED_FUNC(timerObj);
    int n_max_nodes_in_level = static_cast<int>(pow(2, level));
    int nid_offset = static_cast<int>(pow(2, level) - 1);
    int n_column = columns.n_column;
    int n_partition = n_column * n_max_nodes_in_level;
    int nnz = columns.nnz;
    int n_block = std::min((nnz / n_column - 1) / 256 + 1, 32 * 56);

    LOG(TRACE) << "start finding split";

    //find the best split locally
    {
        using namespace thrust;

        //calculate split information for each split
        int n_split;
        SyncArray<GHPair> gh_prefix_sum(nnz);
        SyncArray<GHPair> missing_gh(n_partition);
        SyncArray<int_float> rle_key(nnz);
        if (nnz * 4 > 1.5 * (1 << 30)) rle_key.resize(int(nnz * 0.1));
        auto rle_pid_data = make_transform_iterator(rle_key.device_data(),
                                                    [=]__device__(int_float key) { return get<0>(key); });
        auto rle_fval_data = make_transform_iterator(rle_key.device_data(),
                                                     [=]__device__(int_float key) { return get<1>(key); });
        {

            //gather g/h pairs and do prefix sum
            {
                SyncArray<int> fvid2pid(nnz);
                SyncArray<int> fvid_new2old(nnz);
                {
                    TIMED_SCOPE(timerObj, "find_split - data partitioning");
                    {
                        //input
                        auto *nid_data = stats.nid.device_data();
                        const int *iid_data = columns.csc_row_idx.device_data();

                        LOG(TRACE) << "after using v_stats and columns";
                        //output
                        int *fvid2pid_data = fvid2pid.device_data();
                        device_loop_2d(
                                n_column, columns.csc_col_ptr.device_data(),
                                [=]__device__(int col_id, int fvid) {
                            //feature value id -> instance id -> node id
                            int nid = nid_data[iid_data[fvid]];
                            int pid;
                            //if this node is leaf node, move it to the end
                            if (nid < nid_offset) pid = INT_MAX;//todo negative
                            else pid = col_id * n_max_nodes_in_level + nid - nid_offset;
                            fvid2pid_data[fvid] = pid;
                        },
                        n_block);
                        hipDeviceSynchronize();
                        LOG(DEBUG) << "fvid2pid " << fvid2pid;
                    }

                    //get feature value id mapping for partition, new -> old
                    {
//                    TIMED_SCOPE(timerObj, "fvid_new2old");
                        sequence(cuda::par, fvid_new2old.device_data(), fvid_new2old.device_end(), 0);

                        //using prefix sum memory for temporary storage
                        cub_sort_by_key(fvid2pid, fvid_new2old, -1, true, (void *) gh_prefix_sum.device_data());
                        LOG(DEBUG) << "sorted fvid2pid " << fvid2pid;
                        LOG(DEBUG) << "fvid_new2old " << fvid_new2old;
                    }
                    hipDeviceSynchronize();
                }

                //do prefix sum
                {
                    TIMED_SCOPE(timerObj, "find_split - RLE compression");
                    //same feature value in the same part has the same key.
                    auto key_iter = make_zip_iterator(
                            make_tuple(
                                    fvid2pid.device_data(),
                                    make_permutation_iterator(
                                            columns.csc_val.device_data(),
                                            fvid_new2old.device_data())));//use fvid_new2old to access csc_val
                    n_split = reduce_by_key(
                            cuda::par,
                            key_iter, key_iter + nnz,
                            make_permutation_iterator(                   //ins id -> gh pair
                                    stats.gh_pair.device_data(),
                                    make_permutation_iterator(                 //old fvid -> ins id
                                            columns.csc_row_idx.device_data(),
                                            fvid_new2old.device_data())),             //new fvid -> old fvid
                            rle_key.device_data(),
                            gh_prefix_sum.device_data()
                    ).first - rle_key.device_data();
                    CHECK_LE(n_split, rle_key.size());
                    LOG(DEBUG) << "RLE ratio = " << (float) n_split / nnz;

                    //prefix sum
                    inclusive_scan_by_key(
                            cuda::par,
                            rle_pid_data, rle_pid_data + n_split,
                            gh_prefix_sum.device_data(),
                            gh_prefix_sum.device_data());
                    LOG(DEBUG) << "gh prefix sum = " << gh_prefix_sum;
                    hipDeviceSynchronize();
                }
            }
        }

        //calculate missing value for each partition
        {
            TIMED_SCOPE(timerObj, "find _split - calculate missing value");
            SyncArray<int> pid_ptr(n_partition + 1);
            counting_iterator<int> search_begin(0);
            upper_bound(cuda::par, rle_pid_data, rle_pid_data + n_split, search_begin,
                        search_begin + n_partition, pid_ptr.device_data() + 1);
            LOG(DEBUG) << "pid_ptr = " << pid_ptr;

            auto pid_ptr_data = pid_ptr.device_data();
            auto rle_key_data = rle_key.device_data();
            float_type rt_eps = param.rt_eps;
            {
                SyncArray<float_type> fval(nnz);
                auto fval_data = fval.device_data();
                device_loop(n_split, [=]__device__(int i) {
                    fval_data[i] = rle_fval_data[i];
                });
                device_loop(n_split, [=]__device__(int i) {
                    int pid = rle_pid_data[i];
                    if (pid == INT_MAX) return;
                    float_type f = fval_data[i];
                    if ((pid_ptr_data[pid + 1] - 1) == i)//the last RLE
                        //using "get" to get a modifiable lvalue
                        get<1>(rle_key_data[i]) = (f - fabsf(fval_data[pid_ptr_data[pid]]) - rt_eps);
                    else
                        //FIXME read/write collision
                        get<1>(rle_key_data[i]) = (f + fval_data[i + 1]) * 0.5f;
                });
            }

            const auto gh_prefix_sum_data = gh_prefix_sum.device_data();
            const auto node_data = tree.nodes.device_data();
            auto missing_gh_data = missing_gh.device_data();
            device_loop(n_partition, [=]__device__(int pid) {
                int nid = pid % n_max_nodes_in_level + nid_offset;
                if (pid_ptr_data[pid + 1] != pid_ptr_data[pid])
                    missing_gh_data[pid] =
                            node_data[nid].sum_gh_pair - gh_prefix_sum_data[pid_ptr_data[pid + 1] - 1];
            });
            LOG(DEBUG) << "missing gh = " << missing_gh;
            hipDeviceSynchronize();
        }

        //calculate gain of each split
        SyncArray<float_type> gain(nnz);
        {
            TIMED_SCOPE(timerObj, "find_split - calculate gain");
            auto compute_gain = []__device__(GHPair father, GHPair lch, GHPair rch, float_type min_child_weight,
                    float_type lambda) -> float_type {
                    if (lch.h >= min_child_weight && rch.h >= min_child_weight)
                    return (lch.g * lch.g) / (lch.h + lambda) + (rch.g * rch.g) / (rch.h + lambda) -
            (father.g * father.g) / (father.h + lambda);
                    else
                    return 0;
            };
            const Tree::TreeNode *nodes_data = tree.nodes.device_data();
            GHPair *gh_prefix_sum_data = gh_prefix_sum.device_data();
            float_type *gain_data = gain.device_data();
            const auto missing_gh_data = missing_gh.device_data();
            auto ignored_set_data = ignored_set.device_data();
            //for lambda expression
            float_type mcw = param.min_child_weight;
            float_type l = param.lambda;
            device_loop(n_split, [=]__device__(int i) {
                int pid = rle_pid_data[i];
                int nid0 = pid % n_max_nodes_in_level;
                int fid = pid / n_max_nodes_in_level;
                int nid = nid0 + nid_offset;
                if (pid != INT_MAX && !ignored_set_data[fid]) {
                    GHPair father_gh = nodes_data[nid].sum_gh_pair;
                    GHPair p_missing_gh = missing_gh_data[pid];
                    GHPair rch_gh = gh_prefix_sum_data[i];
                    float_type default_to_left_gain = max(0.f,
                                                          compute_gain(father_gh, father_gh - rch_gh, rch_gh, mcw, l));
                    rch_gh = rch_gh + p_missing_gh;
                    float_type default_to_right_gain = max(0.f,
                                                           compute_gain(father_gh, father_gh - rch_gh, rch_gh, mcw, l));
                    if (default_to_left_gain > default_to_right_gain)
                        gain_data[i] = default_to_left_gain;
                    else
                        gain_data[i] = -default_to_right_gain;//negative means default split to right
                } else gain_data[i] = 0;
            });
            LOG(DEBUG) << "gain = " << gain;
            hipDeviceSynchronize();
        }

        //get best gain and the index of best gain for each feature and each node
        SyncArray<int_float> best_idx_gain(n_partition);
        int n_nodes_in_level;
        {
            TIMED_SCOPE(timerObj, "find_split - get best gain");
            auto arg_abs_max = []__device__(const int_float &a, const int_float &b) {
                if (fabsf(get<1>(a)) == fabsf(get<1>(b)))
                    return get<0>(a) < get<0>(b) ? a : b;
                else
                    return fabsf(get<1>(a)) > fabsf(get<1>(b)) ? a : b;
            };

            //reduce to get best split of each node for this feature
            SyncArray<int> feature_nodes_pid(n_partition);
            int n_feature_with_nodes = reduce_by_key(
                    cuda::par,
                    rle_pid_data, rle_pid_data + n_split,
                    make_zip_iterator(make_tuple(counting_iterator<int>(0), gain.device_data())),
                    feature_nodes_pid.device_data(),
                    best_idx_gain.device_data(),
                    thrust::equal_to<int>(),
                    arg_abs_max).second - best_idx_gain.device_data();

            LOG(DEBUG) << "aaa = " << n_feature_with_nodes;
            LOG(DEBUG) << "f n pid" << feature_nodes_pid;
            LOG(DEBUG) << "best rank & gain = " << best_idx_gain;

            auto feature_nodes_pid_data = feature_nodes_pid.device_data();
            device_loop(n_feature_with_nodes, [=]__device__(int i) {
                feature_nodes_pid_data[i] = feature_nodes_pid_data[i] % n_max_nodes_in_level;
            });
            LOG(DEBUG) << "f n pid" << feature_nodes_pid;
            cub_sort_by_key(feature_nodes_pid, best_idx_gain, n_feature_with_nodes);
            LOG(DEBUG) << "f n pid" << feature_nodes_pid;
            LOG(DEBUG) << "best rank & gain = " << best_idx_gain;
            n_nodes_in_level = reduce_by_key(
                    cuda::par,
                    feature_nodes_pid.device_data(), feature_nodes_pid.device_data() + n_feature_with_nodes,
                    best_idx_gain.device_data(),
                    make_discard_iterator(),
                    best_idx_gain.device_data(),
                    thrust::equal_to<int>(),
                    arg_abs_max
            ).second - best_idx_gain.device_data();
            LOG(DEBUG) << "#nodes in level = " << n_nodes_in_level;
            LOG(DEBUG) << "best rank & gain = " << best_idx_gain;
            hipDeviceSynchronize();
        }

        //get split points
        const int_float *best_idx_gain_data = best_idx_gain.device_data();
        GHPair *gh_prefix_sum_data = gh_prefix_sum.device_data();
        const auto missing_gh_data = missing_gh.device_data();

        sp.resize(n_max_nodes_in_level);
        auto sp_data = sp.device_data();

        int column_offset = columns.column_offset;
        device_loop(n_max_nodes_in_level, [=]__device__(int i) {
            sp_data[i].nid = -1;
        });
        device_loop(n_nodes_in_level, [=]__device__(int i) {
            int_float bst = best_idx_gain_data[i];
            float_type best_split_gain = get<1>(bst);
            int split_index = get<0>(bst);
            int pid = rle_pid_data[split_index];
            if (pid != INT_MAX) {
                int nid0 = pid % n_max_nodes_in_level;
                sp_data[nid0].nid = nid0 + nid_offset;
                sp_data[nid0].split_fea_id = pid / n_max_nodes_in_level + column_offset;
                sp_data[nid0].gain = fabsf(best_split_gain);
                sp_data[nid0].fval = rle_fval_data[split_index];
                sp_data[nid0].fea_missing_gh = missing_gh_data[pid];
                sp_data[nid0].default_right = best_split_gain < 0;
                sp_data[nid0].rch_sum_gh = gh_prefix_sum_data[split_index];
            }
        });
    }

    LOG(DEBUG) << "split points (gain/fea_id/nid): " << sp;
    hipDeviceSynchronize();
}

void ExactTreeBuilder::InternalShard::update_ins2node_id() {
    SyncArray<bool> has_splittable(1);
    //set new node id for each instance
    {
//        TIMED_SCOPE(timerObj, "get new node id");
        auto nid_data = stats.nid.device_data();
        const int *iid_data = columns.csc_row_idx.device_data();
        const Tree::TreeNode *nodes_data = tree.nodes.device_data();
        const int *col_ptr_data = columns.csc_col_ptr.device_data();
        const float_type *f_val_data = columns.csc_val.device_data();
        has_splittable.host_data()[0] = false;
        bool *h_s_data = has_splittable.device_data();
        int column_offset = columns.column_offset;

        int n_column = columns.n_column;
        int nnz = columns.nnz;
        int n_block = std::min((nnz / n_column - 1) / 256 + 1, 32 * 56);

        LOG(TRACE) << "update ins2node id for each fval";
        device_loop_2d(n_column, col_ptr_data,
                       [=]__device__(int col_id, int fvid) {
            //feature value id -> instance id
            int iid = iid_data[fvid];
            //instance id -> node id
            int nid = nid_data[iid];
            //node id -> node
            const Tree::TreeNode &node = nodes_data[nid];
            //if the node splits on this feature
            if (node.splittable() && node.split_feature_id == col_id + column_offset) {
                h_s_data[0] = true;
                if (f_val_data[fvid] < node.split_value)
                    //goes to left child
                    nid_data[iid] = node.lch_index;
                else
                    //right child
                    nid_data[iid] = node.rch_index;
            }
        }, n_block);

    }
    LOG(DEBUG) << "new tree_id = " << stats.nid;
    has_split = has_splittable.host_data()[0];
}

void ExactTreeBuilder::split_point_all_reduce(int depth, vector<InternalShard> &shards) {
    TIMED_FUNC(timerObj);
    //get global best split of each node
    int n_nodes_in_level = 1 << depth;//2^i
    int nid_offset = (1 << depth) - 1;//2^i - 1
    auto global_sp_data = shards.front().sp.host_data();
    vector<bool> active_sp(n_nodes_in_level);

    for (int device_id = 0; device_id < param.n_device; device_id++) {
        auto local_sp_data = shards[device_id].sp.host_data();
        for (int j = 0; j < shards[device_id].sp.size(); j++) {
            int sp_nid = local_sp_data[j].nid;
            if (sp_nid == -1) continue;
            int global_pos = sp_nid - nid_offset;
            if (!active_sp[global_pos])
                global_sp_data[global_pos] = local_sp_data[j];
            else
                global_sp_data[global_pos] = (global_sp_data[global_pos].gain >= local_sp_data[j].gain)
                                             ?
                                             global_sp_data[global_pos] : local_sp_data[j];
            active_sp[global_pos] = true;
        }
    }
    //set inactive sp
    for (int n = 0; n < n_nodes_in_level; n++) {
        if (!active_sp[n])
            global_sp_data[n].nid = -1;
    }
    for_each_shard(shards, [&](InternalShard &shard) {
        shard.sp.copy_from(shards.front().sp);
    });
    LOG(DEBUG) << "global best split point = " << shards.front().sp;
}

void ExactTreeBuilder::ins2node_id_all_reduce(vector<InternalShard> &shards, int depth) {
    //get global ins2node id
    {
        SyncArray<int> local_ins2node_id(shards.front().stats.n_instances);
        auto local_ins2node_id_data = local_ins2node_id.device_data();
        auto global_ins2node_id_data = shards.front().stats.nid.device_data();
        for (int d = 1; d < param.n_device; d++) {
            local_ins2node_id.copy_from(shards[d].stats.nid);
            device_loop(shards.front().stats.n_instances, [=]__device__(int i) {
                global_ins2node_id_data[i] = (global_ins2node_id_data[i] > local_ins2node_id_data[i]) ?
                                             global_ins2node_id_data[i] : local_ins2node_id_data[i];
            });
        }
    }

    //processing missing value
    {
        int n_nodes_in_level = 1 << depth;//2^i
        int nid_offset = (1 << depth) - 1;//2^i - 1
//        TIMED_SCOPE(timerObj, "process missing value");
        LOG(TRACE) << "update ins2node id for each missing fval";
        auto global_ins2node_id_data = shards.front().stats.nid.device_data();//essential
        auto nodes_data = shards.front().tree.nodes.device_data();//already broadcast above
        device_loop(shards.front().stats.n_instances, [=]__device__(int iid) {
            int nid = global_ins2node_id_data[iid];
            //if the instance is not on leaf node and not goes down
            if (nodes_data[nid].splittable() && nid < nid_offset + n_nodes_in_level) {
                //let the instance goes down
                const Tree::TreeNode &node = nodes_data[nid];
                if (node.default_right)
                    global_ins2node_id_data[iid] = node.rch_index;
                else
                    global_ins2node_id_data[iid] = node.lch_index;
            }
        });
        LOG(DEBUG) << "new nid = " << shards.front().stats.nid;
    }

    //broadcast ins2node id
    for_each_shard(shards, [&](InternalShard &shard) {
        shard.stats.nid.copy_from(shards.front().stats.nid);
    });
}

const MSyncArray<float_type>& ExactTreeBuilder::get_y_predict() {
    return y_predict;
}

void ExactTreeBuilder::init(const DataSet &dataset, const GBMParam &param) {
    FunctionBuilder::init(dataset, param);
    //TODO refactor

    this->param = param;
    //init shards
    int n_device = param.n_device;
    shards = vector<InternalShard>(n_device);
    vector<std::unique_ptr<SparseColumns>> v_columns(param.n_device);
    for (int i = 0; i < param.n_device; ++i) {
        v_columns[i].reset(&shards[i].columns);
        shards[i].rank = i;
    }
    SparseColumns columns;
    columns.from_dataset(dataset);
    columns.to_multi_devices(v_columns);
    y_predict = MSyncArray<float_type>(param.n_device);
    for_each_shard(shards, [&](InternalShard &shard) {
        int n_instances = shard.columns.n_row;
        shard.stats.resize(n_instances);
        shard.stats.y_predict = SyncArray<float_type>(param.num_class * n_instances);
        shard.param = param;

        shard.ignored_set.resize(shard.columns.n_column);
        y_predict[shard.rank] = SyncArray<float_type>(shard.stats.y_predict.size());
        y_predict[shard.rank].set_device_data(shard.stats.y_predict.device_data());
    });

    for (int i = 0; i < param.n_device; ++i) {
        v_columns[i].release();
    }
    SyncMem::clear_cache();
}

vector<Tree> ExactTreeBuilder::build_approximate(const MSyncArray<GHPair> &gradients) {
    vector<Tree> trees(param.num_class);
    TIMED_FUNC(timerObj);
    for (int k = 0; k < param.num_class; ++k) {
        Tree &tree = trees[k];
        for_each_shard(shards, [&](InternalShard &shard) {
            shard.stats.gh_pair.set_device_data(const_cast<GHPair *>(gradients[shard.rank].device_data() + k * shard.stats.n_instances));
            shard.stats.reset_nid();//set nid of all the instances to 0
            //todo multi-class bagging, column sampling
            shard.column_sampling();//RF uses this, and may be used by GBDTs
            if (param.bagging) shard.stats.do_bagging();//obtain a bag of instances
            shard.tree.init(shard.stats, param);//init root node, reserve memory, etc.
        });
        for (int level = 0; level < param.depth; ++level) {
            for_each_shard(shards, [&](InternalShard &shard) {
                shard.find_split(level);
            });
            split_point_all_reduce(level, shards);
            {
                TIMED_SCOPE(timerObj, "apply sp");
                for_each_shard(shards, [&]( InternalShard &shard) {
                    shard.update_tree();
                    shard.update_ins2node_id();
                });
                {
                    LOG(TRACE) << "gathering ins2node id";
                    //get final result of the reset instance id to node id
                    bool has_split = false;
                    for (int d = 0; d < param.n_device; d++) {
                        has_split |= shards[d].has_split;
                    }
                    if (!has_split) {
                        LOG(INFO) << "no splittable nodes, stop";
                        break;
                    }
                }
                ins2node_id_all_reduce(shards, level);
            }
        }
        for_each_shard(shards, [&](Shard &shard) {
            shard.tree.prune_self(param.gamma);
            shard.predict_in_training(k);
        });
        tree.nodes.resize(shards.front().tree.nodes.size());
        tree.nodes.copy_from(shards.front().tree.nodes);
    }
    return trees;
}
