//
// Created by ss on 19-1-13.
//
#include "thundergbm/metric/metric.h"
#include "thundergbm/metric/pointwise_metric.h"
#include "thundergbm/metric/ranking_metric.h"
#include "thundergbm/metric/multiclass_metric.h"

Metric *Metric::create(string name) {
    if (name == "map") return new MAP;
    if (name == "rmse") return new RMSE;
    if (name == "ndcg") return new NDCG;
    if (name == "macc") return new MulticlassAccuracy;
    LOG(FATAL) << "unknown metric " << name;
    return nullptr;
}

void Metric::configure(const GBMParam &param, const DataSet &dataset) {
    y.resize(dataset.y.size());
    y.copy_from(dataset.y.data(), dataset.n_instances());
}
