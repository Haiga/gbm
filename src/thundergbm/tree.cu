#include "hip/hip_runtime.h"
//
// Created by jiashuai on 18-1-18.
//
#include "thundergbm/tree.h"
#include "thundergbm/util/device_lambda.cuh"
#include "thrust/reduce.h"

void Tree::init(const InsStat &stats, const GBMParam &param) {
//    TIMED_FUNC(timerObj);
    int n_max_nodes = static_cast<int>(pow(2, param.depth + 1) - 1);
    nodes = SyncArray<TreeNode>(n_max_nodes);
    auto node_data = nodes.device_data();
    device_loop(n_max_nodes, [=]__device__(int i) {
        node_data[i].final_id = i;
        node_data[i].split_feature_id = -1;
        node_data[i].is_valid = false;
        node_data[i].parent_index = i == 0 ? -1 : (i - 1) / 2;
        if (i < n_max_nodes / 2) {
            node_data[i].is_leaf = false;
            node_data[i].lch_index = i * 2 + 1;
            node_data[i].rch_index = i * 2 + 2;
        } else {
            //leaf nodes
            node_data[i].is_leaf = true;
            node_data[i].lch_index = -1;
            node_data[i].rch_index = -1;
        }
    });

    //init root node
    GHPair sum_gh = thrust::reduce(thrust::cuda::par, stats.gh_pair.device_data(), stats.gh_pair.device_end());
    float_type lambda = param.lambda;
    device_loop<1, 1>(1, [=]__device__(int i) {
        Tree::TreeNode &root_node = node_data[0];
        root_node.sum_gh_pair = sum_gh;
        root_node.is_valid = true;
        root_node.calc_weight(lambda);
    });
}

string Tree::dump(int depth) const {
    string s("\n");
    preorder_traversal(0, depth, 0, s);
    return s;
}

void Tree::preorder_traversal(int nid, int max_depth, int depth, string &s) const {
    if(nid == -1)//child of leaf node
        return;
    const TreeNode &node = nodes.host_data()[nid];
    const TreeNode *node_data = nodes.host_data();
    if (node.is_valid && !node.is_pruned) {
        s = s + string(static_cast<unsigned long>(depth), '\t');

        if(node.is_leaf){
            s = s + string_format("%d:leaf=%.6g\n", node.final_id, node.base_weight);
        }
        else {
            int lch_final_id = node_data[node.lch_index].final_id;
            int rch_final_id = node_data[node.rch_index].final_id;
            string str_inter_node = string_format("%d:[f%d<%.6g] yes=%d,no=%d,missing=%d\n", node.final_id,
                                                  node.split_feature_id + 1,
                                                  node.split_value, lch_final_id, rch_final_id,
                                                  node.default_right == 0 ? lch_final_id : rch_final_id);
            s = s + str_inter_node;
        }
//             string_format("%d:[f%d<%.6g], weight=%f, gain=%f, dr=%d\n", node.final_id, node.split_feature_id + 1,
//                           node.split_value,
//                           node.base_weight, node.gain, node.default_right));
    }
    if (depth < max_depth) {
        preorder_traversal(node.lch_index, max_depth, depth + 1, s);
        preorder_traversal(node.rch_index, max_depth, depth + 1, s);
    }
}

std::ostream &operator<<(std::ostream &os, const Tree::TreeNode &node) {
    os << string_format("\nnid:%d,l:%d,v:%d,split_feature_id:%d,f:%f,gain:%f,r:%d,w:%f,", node.final_id, node.is_leaf,
                        node.is_valid,
                        node.split_feature_id, node.split_value, node.gain, node.default_right, node.base_weight);
    os << "g/h:" << node.sum_gh_pair;
    return os;
}

void Tree::reorder_nid() {
    int nid = 0;
    Tree::TreeNode *nodes_data = nodes.host_data();
    for (int i = 0; i < nodes.size(); ++i) {
        if (nodes_data[i].is_valid && !nodes_data[i].is_pruned) {
            nodes_data[i].final_id = nid;
            nid++;
        }
    }
}

int Tree::try_prune_leaf(int nid, int np, float_type gamma, vector<int> &leaf_child_count) {
    Tree::TreeNode *nodes_data = nodes.host_data();
    int p_nid = nodes_data[nid].parent_index;
    if (p_nid == -1) return np;// is root
    Tree::TreeNode &p_node = nodes_data[p_nid];
    Tree::TreeNode &lch = nodes_data[p_node.lch_index];
    Tree::TreeNode &rch = nodes_data[p_node.rch_index];
    leaf_child_count[p_nid]++;
    if (leaf_child_count[p_nid] >= 2 && p_node.gain < gamma) {
        //do pruning
        //delete two children
        CHECK(lch.is_leaf);
        CHECK(rch.is_leaf);
        lch.is_pruned = true;
        rch.is_pruned = true;
        //make parent to leaf
        p_node.is_leaf = true;
        return try_prune_leaf(p_nid, np + 2, gamma, leaf_child_count);
    } else return np;
}

void Tree::prune_self(float_type gamma) {
    vector<int> leaf_child_count(nodes.size(), 0);
    Tree::TreeNode *nodes_data = nodes.host_data();
    int n_pruned = 0;
    for (int i = 0; i < nodes.size(); ++i) {
        if (nodes_data[i].is_leaf && nodes_data[i].is_valid) {
            n_pruned = try_prune_leaf(i, n_pruned, gamma, leaf_child_count);
        }
    }
    LOG(DEBUG) << string_format("%d nodes are pruned", n_pruned);
    reorder_nid();
}
