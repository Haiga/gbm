#include "hip/hip_runtime.h"
//
// Created by shijiashuai on 5/7/18.
//
#include "thundergbm/ins_stat.h"
//#include "thrust/reduce.h"
#include "thrust/random.h"
#include "thundergbm/util/device_lambda.cuh"

void InsStat::resize(size_t n_instances) {
    this->n_instances = n_instances;
    gh_pair.resize(n_instances);
    nid.resize(n_instances);
    y.resize(n_instances);
//    y_predict.resize(n_instances);
}

void InsStat::update_gradient() {
    LOG(DEBUG) << y_predict;
    obj->get_gradient(y, y_predict, gh_pair);
}

void InsStat::reset_nid() {
    auto nid_data = nid.device_data();
    device_loop(n_instances, [=]__device__(int i) {
        nid_data[i] = 0;
    });
}

void InsStat::do_bagging() {
    LOG(TRACE) << "do bagging";
    using namespace thrust;
    SyncArray<int> idx(n_instances);
    auto idx_data = idx.device_data();
    int n_instances = this->n_instances;
    int seed = std::rand();//TODO add a global random generator class
    device_loop(n_instances, [=]__device__(int i) {
        default_random_engine rng(seed);
        uniform_int_distribution<int> uniform_dist(0, n_instances - 1);
        rng.discard(i);
        idx_data[i] = uniform_dist(rng);
    });
    SyncArray<int> ins_count(n_instances);
    auto ins_count_data = ins_count.device_data();
    device_loop(n_instances, [=]__device__(int i) {
        int ins_id = idx_data[i];
        atomicAdd(ins_count_data + ins_id, 1);
    });
//    gh_pair.copy_from(gh_pair_backup);
    auto gh_data = gh_pair.device_data();
    //FIXME synchronize between shards
    device_loop(n_instances, [=]__device__(int i) {
        gh_data[i].g = gh_data[i].g * ins_count_data[i];
        gh_data[i].h = gh_data[i].h * ins_count_data[i];
    });
}

