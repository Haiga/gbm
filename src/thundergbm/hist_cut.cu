#include "hip/hip_runtime.h"
//
// Created by qinbin on 2018/5/9.
//

#include "thundergbm/hist_cut.h"
#include "thundergbm/quantile_sketch.h"
#include "thundergbm/syncarray.h"
#include <sstream>
#include <omp.h>
#include <thundergbm/hist_cut.h>

#include "thundergbm/util/device_lambda.cuh"
#include "thrust/unique.h"

void HistCut::get_cut_points(SparseColumns &columns, InsStat &stats, int max_num_bins, int n_instances) {
    LOG(TRACE) << "get cut points";
    LOG(DEBUG) << "val = " << columns.csc_val;
    LOG(DEBUG) << "idx = " << columns.csc_row_idx;
    LOG(DEBUG) << "ptr = " << columns.csc_col_ptr;
    int n_features = columns.n_column;
//    std::cout<<"n_featrues:"<<n_features<<std::endl;
    vector<quanSketch> sketchs(n_features);
    //kFactor times more cut point candidates are considered when building the summary.
    const int kFactor = 8;
    for (int i = 0; i < n_features; i++) {
        sketchs[i].Init(n_instances, 1.0 / (max_num_bins * kFactor));
    }
    float_type *val_ptr = columns.csc_val.host_data();
    int *row_ptr = columns.csc_row_idx.host_data();
    int *col_ptr = columns.csc_col_ptr.host_data();
    auto stat_gh_ptr = stats.gh_pair.host_data();
//	std::cout<<"before add"<<std::endl;
#pragma omp parallel for
    for (int i = 0; i < columns.csc_col_ptr.size() - 1; i++) {
        for (int j = col_ptr[i + 1] - 1; j >= col_ptr[i]; j--) {
            float_type val = val_ptr[j];
            float_type weight = stat_gh_ptr[row_ptr[j]].h;
            sketchs[i].Add(val, weight);
        }
    }
//    std::cout<<"after add"<<std::endl;
    vector<summary> n_summary(n_features);
//    summary n_summary[n_features];
//	std::cout<<"before prune"<<std::endl;
#pragma omp parallel for
    for (int i = 0; i < n_features; i++) {
        summary ts;
        sketchs[i].GetSummary(ts);
        n_summary[i].Reserve(max_num_bins * kFactor);
        n_summary[i].Prune(ts, max_num_bins * kFactor);
    }
    int nthread = omp_get_max_threads();
//    LOG(DEBUG)<<"nthread = " << nthread;
    vector<vector<float_type>> cut_points_local;
    cut_points_local.resize(n_features);
    vector<int> cut_points_size(n_features);
    for (int i = 0; i < n_features; i++)
        cut_points_local[i].resize(max_num_bins);
#pragma omp parallel num_threads(nthread)
    {
        int tid = omp_get_thread_num();
//        LOG(DEBUG)<<"tid = "<< tid;
        int nstep = (n_features + nthread - 1) / nthread;
        int sbegin = std::min(tid * nstep, n_features);
        int send = std::min((tid + 1) * nstep, n_features);
        for (int i = sbegin; i < send; i++) {
            int k = 0;
            summary ts;
            ts.Reserve(max_num_bins);
            ts.Prune(n_summary[i], max_num_bins);
            if (ts.entry_size == 0) {
                cut_points_size[i] = 0;
                continue;
            }
            float_type min_val = ts.entries[0].val;

            cut_points_local[i][k++] = min_val - (fabsf(min_val) + 1e-5);

            if (ts.entry_size > 1 && ts.entry_size <= 16) {
                cut_points_local[i][k++] = (ts.entries[0].val + ts.entries[1].val) / 2;
                for (int j = 2; j < ts.entry_size; j++) {
                    float_type mid = (ts.entries[j - 1].val + ts.entries[j].val) / 2;
                    if (mid > cut_points_local[i][k - 1]) {
                        cut_points_local[i][k++] = mid;
                    }
                }
            } else {
                if (ts.entry_size > 1)
                    cut_points_local[i][k++] = ts.entries[1].val;
                for (int j = 2; j < ts.entry_size; j++) {
                    float_type val = ts.entries[j].val;
                    if (val > cut_points_local[i][k - 1]) {
                        cut_points_local[i][k++] = val;
                    }
                }
            }

            /*
            float_type max_val = ts.entries[ts.entry_size - 1].val;
            if(max_val > 0){
                cut_points_local[i][k++] = max_val*2 + 1e-5;
            }
            else{
                cut_points_local[i][k++] = 1e-5;
            }
            */
            cut_points_size[i] = k;
        }
    }
    for (int i = 0; i < n_features; i++) {
        if (cut_points_size[i] != 0)
            this->cut_points.insert(cut_points.end(), cut_points_local[i].begin(),
                                    cut_points_local[i].begin() + cut_points_size[i]);
    }
    this->row_ptr.push_back(0);
    for (int i = 0; i < n_features; i++) {
        this->row_ptr.push_back(cut_points_size[i] + this->row_ptr.back());
    }
    cut_row_ptr.resize(this->row_ptr.size());
    cut_row_ptr.copy_from(this->row_ptr.data(), this->row_ptr.size());
    cut_points_val.resize(this->cut_points.size());
    auto cut_points_val_ptr = cut_points_val.host_data();
    auto cut_row_ptr_data = cut_row_ptr.host_data();
    for (int i = 0; i < cut_row_ptr.size(); i++) {
        int sum = cut_row_ptr_data[i] + cut_row_ptr_data[i + 1] - 1;
        for (int j = cut_row_ptr_data[i + 1] - 1; j >= cut_row_ptr_data[i]; j--)
            cut_points_val_ptr[j] = this->cut_points[sum - j];
    }
    LOG(DEBUG) << cut_row_ptr;
    LOG(DEBUG) << cut_fid.size();
    cut_fid.resize(cut_points.size());
    auto cut_fid_data = cut_fid.device_data();
    device_loop_2d(n_features, cut_row_ptr.device_data(), [=] __device__(int fid, int i) {
        cut_fid_data[i] = fid;
    });
}

void HistCut::get_cut_points2(SparseColumns &columns, int max_num_bins, int n_instances) {
    int n_column = columns.n_column;
    auto csc_val_data = columns.csc_val.host_data();
    auto csc_col_ptr_data = columns.csc_col_ptr.host_data();
    cut_points.clear();
    row_ptr.clear();
    row_ptr.resize(1, 0);

    //TODO do this on GPU
    for (int fid = 0; fid < n_column; ++fid) {
        int col_start = csc_col_ptr_data[fid];
        int col_len = csc_col_ptr_data[fid + 1] - col_start;
        auto val_data = csc_val_data + col_start;
        vector<float_type> unique_val(col_len);

        int unique_len = thrust::unique_copy(thrust::host, val_data, val_data + col_len, unique_val.data()) - unique_val.data();
        if (unique_len <= max_num_bins) {
            row_ptr.push_back(unique_len + row_ptr.back());
            for (int i = 0; i < unique_len; ++i) {
                cut_points.push_back(unique_val[i]);
            }
        } else {
            row_ptr.push_back(max_num_bins + row_ptr.back());
            for (int i = 0; i < max_num_bins; ++i) {
                cut_points.push_back(unique_val[unique_len / max_num_bins * i]);
            }
        }
    }

    cut_points_val.resize(cut_points.size());
    cut_points_val.copy_from(cut_points.data(), cut_points.size());
    cut_row_ptr.resize(row_ptr.size());
    cut_row_ptr.copy_from(row_ptr.data(), row_ptr.size());
    cut_fid.resize(cut_points.size());
    auto cut_fid_data = cut_fid.device_data();
    device_loop_2d(n_column, cut_row_ptr.device_data(), [=] __device__(int fid, int i) {
        cut_fid_data[i] = fid;
    });
}
